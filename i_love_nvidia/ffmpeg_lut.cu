#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <stdexcept>
#include <algorithm>
#include <hip/hip_runtime.h>

extern "C" {
#include <libavformat/avformat.h>
#include <libavcodec/avcodec.h>
#include <libavutil/imgutils.h>
#include <libavutil/opt.h>
#include <libswscale/swscale.h>
}

// Macro for CUDA error checking.
#define CUDA_CHECK(call)                                            \
    do {                                                            \
        hipError_t err = call;                                     \
        if (err != hipSuccess) {                                   \
            std::cerr << "CUDA Error: " << hipGetErrorString(err)  \
                      << " at " << __FILE__ << ":" << __LINE__       \
                      << std::endl;                                 \
            exit(EXIT_FAILURE);                                     \
        }                                                           \
    } while (0)

// Macro for FFmpeg error checking.
#define AV_CHECK(err) do { \
    if ((err) < 0) { \
        char errbuf[128]; \
        av_strerror((err), errbuf, sizeof(errbuf)); \
        std::cerr << "FFmpeg error: " << errbuf << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
} while(0)

// Device helper: linear interpolation for float3.
__device__ inline float3 lerp(const float3 &a, const float3 &b, float t) {
    return make_float3(a.x + t * (b.x - a.x),
                       a.y + t * (b.y - a.y),
                       a.z + t * (b.z - a.z));
}

// Device helper function to get LUT value
__device__ float3 getLut(int rr, int gg, int bb, const float* d_lut, int lutSize) {
    int index = ((rr * lutSize * lutSize) + (gg * lutSize) + bb) * 3;
    return make_float3(d_lut[index], d_lut[index+1], d_lut[index+2]);
}

// CUDA kernel: applies a 3D LUT via trilinear interpolation.
// d_input and d_output are RGB24 images (width*height*3 bytes).
// d_lut is a flattened LUT array (lutSize^3*3 floats).
__global__ void applyLUTKernel(const unsigned char* d_input, unsigned char* d_output,
                               int width, int height, const float* d_lut, int lutSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;
    int idx = (y * width + x) * 3;
    
    // Check if input is pure white
    if (d_input[idx] == 255 && d_input[idx+1] == 255 && d_input[idx+2] == 255) {
        d_output[idx] = d_output[idx+1] = d_output[idx+2] = 255;
        return;
    }
    
    float r = d_input[idx]   / 255.f;
    float g = d_input[idx+1] / 255.f;
    float b = d_input[idx+2] / 255.f;
    float fr = r * (lutSize - 1);
    float fg = g * (lutSize - 1);
    float fb = b * (lutSize - 1);
    int r0 = floorf(fr), g0 = floorf(fg), b0 = floorf(fb);
    int r1 = min(r0 + 1, lutSize - 1);
    int g1 = min(g0 + 1, lutSize - 1);
    int b1 = min(b0 + 1, lutSize - 1);
    float dr = fr - r0, dg = fg - g0, db = fb - b0;
    
    float3 c000 = getLut(r0, g0, b0, d_lut, lutSize);
    float3 c100 = getLut(r1, g0, b0, d_lut, lutSize);
    float3 c010 = getLut(r0, g1, b0, d_lut, lutSize);
    float3 c001 = getLut(r0, g0, b1, d_lut, lutSize);
    float3 c101 = getLut(r1, g0, b1, d_lut, lutSize);
    float3 c011 = getLut(r0, g1, b1, d_lut, lutSize);
    float3 c110 = getLut(r1, g1, b0, d_lut, lutSize);
    float3 c111 = getLut(r1, g1, b1, d_lut, lutSize);
    
    float3 c00 = lerp(c000, c100, dr);
    float3 c01 = lerp(c001, c101, dr);
    float3 c10 = lerp(c010, c110, dr);
    float3 c11 = lerp(c011, c111, dr);
    float3 c0 = lerp(c00, c10, dg);
    float3 c1 = lerp(c01, c11, dg);
    float3 c  = lerp(c0,  c1,  db);

    // Ensure we preserve full range output
    d_output[idx]   = static_cast<unsigned char>(fminf(fmaxf(c.x * 255.f + 0.5f, 0.f), 255.f));
    d_output[idx+1] = static_cast<unsigned char>(fminf(fmaxf(c.y * 255.f + 0.5f, 0.f), 255.f));
    d_output[idx+2] = static_cast<unsigned char>(fminf(fmaxf(c.z * 255.f + 0.5f, 0.f), 255.f));
}

// Structure to hold a 3D LUT.
struct LUT3D {
    int size;
    std::vector<float> data; // Contains size^3 * 3 entries.
};

// Parses a .lut file expecting a header line "LUT_3D_SIZE N" followed by N^3 lines with three floats.
LUT3D loadLUT(const std::string& filename) {
    std::ifstream file(filename);
    if (!file.is_open())
        throw std::runtime_error("Failed to open LUT file: " + filename);
    LUT3D lut;
    std::string line;
    while (std::getline(file, line)) {
        if (line.empty() || line[0] == '#') continue;
        std::istringstream iss(line);
        std::string token;
        iss >> token;
        if (token == "LUT_3D_SIZE") {
            if (!(iss >> lut.size) || lut.size <= 0)
                throw std::runtime_error("Invalid LUT size in header.");
            break;
        }
    }
    if (lut.size <= 0)
        throw std::runtime_error("LUT size not specified in LUT file.");
    size_t expectedEntries = lut.size * lut.size * lut.size * 3;
    lut.data.reserve(expectedEntries);
    while (std::getline(file, line)) {
        if (line.empty() || line[0] == '#') continue;
        std::istringstream iss(line);
        float r, g, b;
        if (!(iss >> r >> g >> b))
            continue;
        lut.data.push_back(r);
        lut.data.push_back(g);
        lut.data.push_back(b);
    }
    if (lut.data.size() != expectedEntries)
        throw std::runtime_error("LUT file does not contain the expected number of entries.");
    return lut;
}

int main(int argc, char* argv[]) {
    if (argc < 4) {
        std::cerr << "Usage: " << argv[0] << " <lut_file> <input_video> <output_video>" << std::endl;
        return EXIT_FAILURE;
    }
    std::string lutFile = argv[1];
    std::string inputVideo = argv[2];
    std::string outputVideo = argv[3];

    // Initialize all pointers to nullptr
    AVFormatContext* inFmtCtx = nullptr;
    AVFormatContext* outFmtCtx = nullptr;
    AVCodecContext* decCtx = nullptr;
    AVCodecContext* encCtx = nullptr;
    AVFrame* decFrame = nullptr;
    AVFrame* rgbFrame = nullptr;
    AVFrame* encFrame = nullptr;
    AVPacket* packet = nullptr;
    SwsContext* swsCtxToRGB = nullptr;
    SwsContext* swsCtxFromRGB = nullptr;
    unsigned char *d_input = nullptr, *d_output = nullptr;
    float* d_lut = nullptr;

    // Load LUT and copy it to device memory.
    LUT3D lut;
    try {
        lut = loadLUT(lutFile);
    } catch (const std::exception &ex) {
        std::cerr << "LUT loading error: " << ex.what() << std::endl;
        return EXIT_FAILURE;
    }
    size_t lutBytes = lut.data.size() * sizeof(float);
    CUDA_CHECK(hipMalloc(&d_lut, lutBytes));
    CUDA_CHECK(hipMemcpy(d_lut, lut.data.data(), lutBytes, hipMemcpyHostToDevice));

    // Open the input video file.
    int ret = avformat_open_input(&inFmtCtx, inputVideo.c_str(), nullptr, nullptr);
    AV_CHECK(ret);
    ret = avformat_find_stream_info(inFmtCtx, nullptr);
    AV_CHECK(ret);

    // Find the video stream.
    int videoStreamIndex = -1;
    for (unsigned int i = 0; i < inFmtCtx->nb_streams; i++) {
        if (inFmtCtx->streams[i]->codecpar->codec_type == AVMEDIA_TYPE_VIDEO) {
            videoStreamIndex = i;
            break;
        }
    }
    if (videoStreamIndex == -1) {
        std::cerr << "No video stream found." << std::endl;
        return EXIT_FAILURE;
    }
    AVStream* videoStream = inFmtCtx->streams[videoStreamIndex];

    // Open the decoder.
    const AVCodec* decoder = avcodec_find_decoder(videoStream->codecpar->codec_id);
    if (!decoder) {
        std::cerr << "Decoder not found." << std::endl;
        return EXIT_FAILURE;
    }
    decCtx = avcodec_alloc_context3(decoder);
    if (!decCtx) {
        std::cerr << "Failed to allocate decoder context." << std::endl;
        return EXIT_FAILURE;
    }
    ret = avcodec_parameters_to_context(decCtx, videoStream->codecpar);
    AV_CHECK(ret);
    ret = avcodec_open2(decCtx, decoder, nullptr);
    AV_CHECK(ret);

    // Set up a SwsContext to convert the decoded frame to RGB24 (which our CUDA kernel expects).
    swsCtxToRGB = sws_getContext(decCtx->width, decCtx->height, decCtx->pix_fmt,
                                             decCtx->width, decCtx->height, AV_PIX_FMT_RGB24,
                                             SWS_BILINEAR, nullptr, nullptr, nullptr);
    if (!swsCtxToRGB) {
        std::cerr << "Could not initialize sws context for RGB conversion." << std::endl;
        return EXIT_FAILURE;
    }

    // Set up the output file and encoder.
    ret = avformat_alloc_output_context2(&outFmtCtx, nullptr, nullptr, outputVideo.c_str());
    if (!outFmtCtx) {
        std::cerr << "Could not create output context." << std::endl;
        return EXIT_FAILURE;
    }
    const AVCodec* encoder = avcodec_find_encoder(AV_CODEC_ID_H264);
    if (!encoder) {
        std::cerr << "Necessary encoder not found." << std::endl;
        return EXIT_FAILURE;
    }
    AVStream* outStream = avformat_new_stream(outFmtCtx, nullptr);
    if (!outStream) {
        std::cerr << "Failed allocating output stream." << std::endl;
        return EXIT_FAILURE;
    }

    encCtx = avcodec_alloc_context3(encoder);
    if (!encCtx) {
        std::cerr << "Failed to allocate encoder context." << std::endl;
        return EXIT_FAILURE;
    }

    // Set encoder parameters
    encCtx->width = decCtx->width;
    encCtx->height = decCtx->height;
    encCtx->sample_aspect_ratio = decCtx->sample_aspect_ratio;
    encCtx->pix_fmt = AV_PIX_FMT_YUV420P;

    // Set reasonable defaults for x264 encoding
    encCtx->bit_rate = 2000000;  // 2 Mbps
    encCtx->rc_max_rate = 2000000;
    encCtx->rc_min_rate = 2000000;
    encCtx->rc_buffer_size = 4000000;  // 2 seconds worth of data
    encCtx->gop_size = 12;
    encCtx->max_b_frames = 2;
    encCtx->thread_count = 0;  // Let FFmpeg decide thread count
    
    // Set timebase and framerate
    if (videoStream->r_frame_rate.num && videoStream->r_frame_rate.den) {
        encCtx->framerate = videoStream->r_frame_rate;
        encCtx->time_base = av_inv_q(videoStream->r_frame_rate);
    } else {
        encCtx->framerate = (AVRational){25, 1};  // Default to 25 fps
        encCtx->time_base = (AVRational){1, 25};
    }
    
    // Set stream timebase to match input
    outStream->time_base = videoStream->time_base;
    
    if (outFmtCtx->oformat->flags & AVFMT_GLOBALHEADER)
        encCtx->flags |= AV_CODEC_FLAG_GLOBAL_HEADER;

    // Set x264 encoding preset and tune
    AVDictionary *param = nullptr;
    av_dict_set(&param, "preset", "medium", 0);
    av_dict_set(&param, "tune", "film", 0);
    av_dict_set(&param, "profile", "high", 0);
    av_dict_set(&param, "level", "4.0", 0);
    av_dict_set(&param, "rc-lookahead", "20", 0);
    
    ret = avcodec_open2(encCtx, encoder, &param);
    av_dict_free(&param);
    AV_CHECK(ret);

    // Copy encoder parameters to output stream
    ret = avcodec_parameters_from_context(outStream->codecpar, encCtx);
    AV_CHECK(ret);
    
    // Copy relevant stream metadata
    av_dict_copy(&outStream->metadata, videoStream->metadata, 0);

    if (!(outFmtCtx->oformat->flags & AVFMT_NOFILE)) {
        ret = avio_open(&outFmtCtx->pb, outputVideo.c_str(), AVIO_FLAG_WRITE);
        AV_CHECK(ret);
    }
    ret = avformat_write_header(outFmtCtx, nullptr);
    AV_CHECK(ret);

    // Set up a SwsContext to convert RGB24 (processed by CUDA) to the encoder's pixel format.
    swsCtxFromRGB = sws_getContext(decCtx->width, decCtx->height, AV_PIX_FMT_RGB24,
                                               encCtx->width, encCtx->height, encCtx->pix_fmt,
                                               SWS_BILINEAR, nullptr, nullptr, nullptr);
    if (!swsCtxFromRGB) {
        std::cerr << "Could not initialize sws context for encoder conversion." << std::endl;
        return EXIT_FAILURE;
    }

    // Allocate frames for decoding, RGB conversion, and encoding.
    decFrame = av_frame_alloc();
    rgbFrame = av_frame_alloc();
    encFrame = av_frame_alloc();
    if (!decFrame || !rgbFrame || !encFrame) {
        std::cerr << "Could not allocate frames." << std::endl;
        return EXIT_FAILURE;
    }

    // Set up RGB frame
    rgbFrame->format = AV_PIX_FMT_RGB24;
    rgbFrame->width = decCtx->width;
    rgbFrame->height = decCtx->height;
    ret = av_frame_get_buffer(rgbFrame, 32);
    AV_CHECK(ret);

    // Set up encoding frame
    encFrame->format = encCtx->pix_fmt;
    encFrame->width = encCtx->width;
    encFrame->height = encCtx->height;
    ret = av_frame_get_buffer(encFrame, 32);
    AV_CHECK(ret);

    // Allocate CUDA device buffers for the frame.
    int frameBytes = rgbFrame->linesize[0] * rgbFrame->height;
    CUDA_CHECK(hipMalloc(&d_input, frameBytes));
    CUDA_CHECK(hipMalloc(&d_output, frameBytes));

    packet = av_packet_alloc();
    if (!packet) {
        std::cerr << "Failed to allocate packet." << std::endl;
        return EXIT_FAILURE;
    }

    // Main processing loop.
    int frameIndex = 0;
    while (av_read_frame(inFmtCtx, packet) >= 0) {
        if (packet->stream_index == videoStreamIndex) {
            ret = avcodec_send_packet(decCtx, packet);
            if (ret < 0) {
                std::cerr << "Error sending packet for decoding." << std::endl;
                break;
            }
            while (ret >= 0) {
                ret = avcodec_receive_frame(decCtx, decFrame);
                if (ret == AVERROR(EAGAIN) || ret == AVERROR_EOF)
                    break;
                else if (ret < 0) {
                    std::cerr << "Error during decoding." << std::endl;
                    break;
                }

                // Convert the decoded frame to RGB24.
                ret = av_frame_make_writable(rgbFrame);
                if (ret < 0) {
                    std::cerr << "Error making RGB frame writable." << std::endl;
                    break;
                }
                ret = sws_scale(swsCtxToRGB, decFrame->data, decFrame->linesize, 0, decCtx->height,
                          rgbFrame->data, rgbFrame->linesize);
                if (ret < 0) {
                    std::cerr << "Error converting frame to RGB." << std::endl;
                    break;
                }

                // Process the RGB frame with CUDA.
                CUDA_CHECK(hipMemcpy(d_input, rgbFrame->data[0], frameBytes, hipMemcpyHostToDevice));
                dim3 block(16, 16);
                dim3 grid((decCtx->width + block.x - 1) / block.x, (decCtx->height + block.y - 1) / block.y);
                applyLUTKernel<<<grid, block>>>(d_input, d_output, decCtx->width, decCtx->height, d_lut, lut.size);
                CUDA_CHECK(hipDeviceSynchronize());
                CUDA_CHECK(hipMemcpy(rgbFrame->data[0], d_output, frameBytes, hipMemcpyDeviceToHost));

                // Create a new encoding frame for each frame
                AVFrame* newEncFrame = av_frame_alloc();
                if (!newEncFrame) {
                    std::cerr << "Could not allocate new encoding frame." << std::endl;
                    break;
                }
                newEncFrame->format = encCtx->pix_fmt;
                newEncFrame->width = encCtx->width;
                newEncFrame->height = encCtx->height;
                ret = av_frame_get_buffer(newEncFrame, 32);
                if (ret < 0) {
                    std::cerr << "Could not allocate new encoding frame buffer." << std::endl;
                    av_frame_free(&newEncFrame);
                    break;
                }

                // Convert the processed RGB frame to YUV420P for the encoder.
                ret = sws_scale(swsCtxFromRGB, rgbFrame->data, rgbFrame->linesize, 0, decCtx->height,
                          newEncFrame->data, newEncFrame->linesize);
                if (ret < 0) {
                    std::cerr << "Error converting frame to YUV." << std::endl;
                    av_frame_free(&newEncFrame);
                    break;
                }

                // Set frame properties
                newEncFrame->pts = av_rescale_q(decFrame->pts, videoStream->time_base, encCtx->time_base);
                newEncFrame->pkt_dts = AV_NOPTS_VALUE;
                newEncFrame->key_frame = 0;
                newEncFrame->pict_type = AV_PICTURE_TYPE_NONE;

                ret = avcodec_send_frame(encCtx, newEncFrame);
                if (ret < 0) {
                    std::cerr << "Error sending frame to encoder." << std::endl;
                    av_frame_free(&newEncFrame);
                    break;
                }

                // Retrieve and write the encoded packet.
                AVPacket* encPkt = av_packet_alloc();
                if (!encPkt) {
                    std::cerr << "Could not allocate packet." << std::endl;
                    av_frame_free(&newEncFrame);
                    break;
                }

                while (ret >= 0) {
                    ret = avcodec_receive_packet(encCtx, encPkt);
                    if (ret == AVERROR(EAGAIN) || ret == AVERROR_EOF) {
                        break;
                    } else if (ret < 0) {
                        std::cerr << "Error during encoding." << std::endl;
                        break;
                    }
                    
                    // Set packet stream index and rescale timestamps
                    encPkt->stream_index = outStream->index;
                    av_packet_rescale_ts(encPkt, encCtx->time_base, outStream->time_base);
                    
                    // Write the packet
                    ret = av_interleaved_write_frame(outFmtCtx, encPkt);
                    if (ret < 0) {
                        char errbuf[AV_ERROR_MAX_STRING_SIZE];
                        av_strerror(ret, errbuf, sizeof(errbuf));
                        std::cerr << "Error writing frame: " << errbuf << std::endl;
                    }
                }
                av_packet_free(&encPkt);
                av_frame_unref(decFrame);
                av_frame_free(&newEncFrame);
            }
        }
        av_packet_unref(packet);
    }

    // Flush the decoder.
    avcodec_send_packet(decCtx, nullptr);
    while (avcodec_receive_frame(decCtx, decFrame) == 0) {
        // Convert the decoded frame to RGB24.
        ret = av_frame_make_writable(rgbFrame);
        if (ret < 0) {
            std::cerr << "Error making RGB frame writable." << std::endl;
            break;
        }
        ret = sws_scale(swsCtxToRGB, decFrame->data, decFrame->linesize, 0, decCtx->height,
                  rgbFrame->data, rgbFrame->linesize);
        if (ret < 0) {
            std::cerr << "Error converting frame to RGB." << std::endl;
            break;
        }

        // Process the RGB frame with CUDA.
        CUDA_CHECK(hipMemcpy(d_input, rgbFrame->data[0], frameBytes, hipMemcpyHostToDevice));
        dim3 block(16, 16);
        dim3 grid((decCtx->width + block.x - 1) / block.x, (decCtx->height + block.y - 1) / block.y);
        applyLUTKernel<<<grid, block>>>(d_input, d_output, decCtx->width, decCtx->height, d_lut, lut.size);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(rgbFrame->data[0], d_output, frameBytes, hipMemcpyDeviceToHost));

        // Create a new encoding frame
        AVFrame* newEncFrame = av_frame_alloc();
        if (!newEncFrame) {
            std::cerr << "Could not allocate new encoding frame." << std::endl;
            break;
        }
        newEncFrame->format = encCtx->pix_fmt;
        newEncFrame->width = encCtx->width;
        newEncFrame->height = encCtx->height;
        ret = av_frame_get_buffer(newEncFrame, 32);
        if (ret < 0) {
            std::cerr << "Could not allocate new encoding frame buffer." << std::endl;
            av_frame_free(&newEncFrame);
            break;
        }

        // Convert the processed RGB frame to YUV420P for the encoder.
        ret = sws_scale(swsCtxFromRGB, rgbFrame->data, rgbFrame->linesize, 0, decCtx->height,
                  newEncFrame->data, newEncFrame->linesize);
        if (ret < 0) {
            std::cerr << "Error converting frame to YUV." << std::endl;
            av_frame_free(&newEncFrame);
            break;
        }

        // Set frame properties
        newEncFrame->pts = av_rescale_q(decFrame->pts, videoStream->time_base, encCtx->time_base);
        newEncFrame->pkt_dts = AV_NOPTS_VALUE;
        newEncFrame->key_frame = 0;
        newEncFrame->pict_type = AV_PICTURE_TYPE_NONE;

        ret = avcodec_send_frame(encCtx, newEncFrame);
        if (ret < 0) {
            std::cerr << "Error sending frame to encoder." << std::endl;
            av_frame_free(&newEncFrame);
            break;
        }

        // Retrieve and write the encoded packet.
        AVPacket* encPkt = av_packet_alloc();
        if (!encPkt) {
            std::cerr << "Could not allocate packet." << std::endl;
            av_frame_free(&newEncFrame);
            break;
        }

        while (ret >= 0) {
            ret = avcodec_receive_packet(encCtx, encPkt);
            if (ret == AVERROR(EAGAIN) || ret == AVERROR_EOF) {
                break;
            } else if (ret < 0) {
                std::cerr << "Error during encoding." << std::endl;
                break;
            }
            
            // Set packet stream index and rescale timestamps
            encPkt->stream_index = outStream->index;
            av_packet_rescale_ts(encPkt, encCtx->time_base, outStream->time_base);
            
            // Write the packet
            ret = av_interleaved_write_frame(outFmtCtx, encPkt);
            if (ret < 0) {
                char errbuf[AV_ERROR_MAX_STRING_SIZE];
                av_strerror(ret, errbuf, sizeof(errbuf));
                std::cerr << "Error writing frame: " << errbuf << std::endl;
            }
        }
        av_packet_free(&encPkt);
        av_frame_free(&newEncFrame);
        av_frame_unref(decFrame);
    }

    // Flush the encoder
    ret = avcodec_send_frame(encCtx, nullptr);
    if (ret >= 0) {
        while (ret >= 0) {
            AVPacket* encPkt = av_packet_alloc();
            if (!encPkt) {
                std::cerr << "Could not allocate packet." << std::endl;
                break;
            }

            ret = avcodec_receive_packet(encCtx, encPkt);
            if (ret == AVERROR(EAGAIN) || ret == AVERROR_EOF) {
                av_packet_free(&encPkt);
                break;
            } else if (ret < 0) {
                std::cerr << "Error during encoding." << std::endl;
                av_packet_free(&encPkt);
                break;
            }

            // Set packet stream index and rescale timestamps
            encPkt->stream_index = outStream->index;
            av_packet_rescale_ts(encPkt, encCtx->time_base, outStream->time_base);

            // Write the packet
            ret = av_interleaved_write_frame(outFmtCtx, encPkt);
            if (ret < 0) {
                char errbuf[AV_ERROR_MAX_STRING_SIZE];
                av_strerror(ret, errbuf, sizeof(errbuf));
                std::cerr << "Error writing frame: " << errbuf << std::endl;
            }
            av_packet_free(&encPkt);
        }
    }

    // Write the trailer and clean up
    ret = av_write_trailer(outFmtCtx);
    if (ret < 0) {
        char errbuf[AV_ERROR_MAX_STRING_SIZE];
        av_strerror(ret, errbuf, sizeof(errbuf));
        std::cerr << "Error writing trailer: " << errbuf << std::endl;
    }

    // Clean up in reverse order of allocation
    // First, close the output file
    if (outFmtCtx && !(outFmtCtx->oformat->flags & AVFMT_NOFILE)) {
        avio_closep(&outFmtCtx->pb);
    }

    // Free CUDA resources
    if (d_input) {
        hipFree(d_input);
        d_input = nullptr;
    }
    if (d_output) {
        hipFree(d_output);
        d_output = nullptr;
    }
    if (d_lut) {
        hipFree(d_lut);
        d_lut = nullptr;
    }

    // Free SwsContext
    if (swsCtxToRGB) {
        sws_freeContext(swsCtxToRGB);
        swsCtxToRGB = nullptr;
    }
    if (swsCtxFromRGB) {
        sws_freeContext(swsCtxFromRGB);
        swsCtxFromRGB = nullptr;
    }

    // Free frames
    if (decFrame) {
        av_frame_free(&decFrame);
        decFrame = nullptr;
    }
    if (rgbFrame) {
        av_frame_free(&rgbFrame);
        rgbFrame = nullptr;
    }
    if (encFrame) {
        av_frame_free(&encFrame);
        encFrame = nullptr;
    }

    // Free packet
    if (packet) {
        av_packet_free(&packet);
        packet = nullptr;
    }

    // Free codec contexts
    if (decCtx) {
        avcodec_free_context(&decCtx);
        decCtx = nullptr;
    }
    if (encCtx) {
        avcodec_free_context(&encCtx);
        encCtx = nullptr;
    }

    // Close input and free contexts
    if (inFmtCtx) {
        avformat_close_input(&inFmtCtx);
        inFmtCtx = nullptr;
    }
    if (outFmtCtx) {
        avformat_free_context(outFmtCtx);
        outFmtCtx = nullptr;
    }

    std::cout << "Processing completed successfully." << std::endl;
    return EXIT_SUCCESS;
}
